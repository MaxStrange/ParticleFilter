#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <random>
#include <stdio.h>
#include <vector>
#include "device.h"

static float gaussian_noise(float mean, float sigma, std::mt19937 &rng)
{
    std::normal_distribution<float> gaussian(mean, sigma);
    return gaussian(rng);
}

static float probability_of_value_from_bivariate_gaussian(float x, float y, float mean_x, float mean_y, float sigma_x, float sigma_y)
{
    const float rho = 0.0; // cov / (sig1 * sig2); Covariance of two independent random variables is zero.
    float denom = 2.0 * M_PI * sigma_x * sigma_y * sqrt(1.0 - (rho * rho));
    float A = ((x - mean_x) * (x - mean_x)) / (sigma_x * sigma_x);
    float B = ((2.0 * rho * (x - mean_x) * (y - mean_y)) / (sigma_x * sigma_y));
    float C = ((y - mean_y) * (y - mean_y)) / (sigma_y * sigma_y);
    A /= 1000.0;  // For numerical stability
    C /= 1000.0;  // Ditto
    float z = A - B + C;
    float a = (-1.0 * z) / (2.0 * (1.0 - rho * rho));

    return exp(a) / denom;
}

__global__ void kernel_calculate_likelihood(int *particles_x, int *particles_y, float *weights, unsigned int nparticles, float estimate_x, float estimate_y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nparticles)
    {
        float x = (float)particles_x[index];
        float y = (float)particles_y[index];

        const float sigma_x = 2.5;
        const float sigma_y = 2.5;
        float mean_x = estimate_x;
        float mean_y = estimate_y;

        // Compute the probability of getting this x,y combo from a distribution centered at estimate_x, estimte_y.
        const float rho = 0.0; // cov / (sig1 * sig2); Covariance of two independent random variables is zero.
        float denom = 2.0f * M_PI * sigma_x * sigma_y * sqrt(1.0f - (rho * rho));
        float A = ((x - mean_x) * (x - mean_x)) / (sigma_x * sigma_x);
        float B = ((2.0f * rho * (x - mean_x) * (y - mean_y)) / (sigma_x * sigma_y));
        float C = ((y - mean_y) * (y - mean_y)) / (sigma_y * sigma_y);
        A /= 1000.0f;  // For numerical stability
        C /= 1000.0f;  // Ditto
        float z = A - B + C;
        float a = (-1.0f * z) / (2.0f * (1.0f - rho * rho));
        float prob = exp(a) / denom;
        weights[index] = prob;
    }
}

int device_calculate_likelihood(const int *particles_x, const int *particles_y, int estimate_x, int estimate_y, float *weights, unsigned int nparticles, int nthreads_per_block)
{
    hipError_t err;
    int *dev_particles_x = nullptr;
    int *dev_particles_y = nullptr;
    float *dev_weights = nullptr;

    #define CHECK_CUDA_ERR(err) do { if (err != hipSuccess) { err = (hipError_t)__LINE__; goto fail; }} while (0)

    /* Malloc all the device memory we need */
    err = hipMalloc(&dev_particles_x, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_particles_y, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_weights, nparticles * sizeof(float));
    CHECK_CUDA_ERR(err);

    /* Copy arrays onto device */
    err = hipMemcpy(dev_particles_x, particles_x, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_particles_y, particles_y, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_weights, weights, nparticles * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    /* Call the kernel */
    kernel_calculate_likelihood<<<ceil(nparticles / (float)nthreads_per_block), nthreads_per_block>>>(dev_particles_x, dev_particles_y, dev_weights, nparticles, estimate_x, estimate_y);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERR(err);

    /* Copy array back onto host */
    err = hipMemcpy(weights, dev_weights, nparticles * sizeof(float), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    /* Deallocate the device arrays */
    err = hipFree(dev_particles_x);
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_particles_y);
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_weights);
    CHECK_CUDA_ERR(err);

    #undef CHECK_CUDA_ERR

fail:
    assert(err == hipSuccess);
    return (int)err;
}

static void move_particles(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, float *particles_weights, std::mt19937 &rng)
{
    for (unsigned int i = 0; i < nparticles; i++)
    {
        static const float sigma = 2.5;
        float vx = gaussian_noise(estimated_vx, sigma, rng);
        float vy = gaussian_noise(estimated_vy, sigma, rng);
        particles_x[i] += vx;
        particles_y[i] += vy;
        particles_weights[i] = probability_of_value_from_bivariate_gaussian(vx, vy, estimated_vx, estimated_vy, sigma, sigma);
    }
}

static void sort_particles_by_weight_in_place(unsigned int *indices, unsigned int nparticles, float *particles_weights, int *particles_x, int *particles_y)
{
    // Sort the indices
    std::sort(indices, indices + nparticles, SortIndices(particles_weights));

    // Make copies of the three arrays (gross)
    int *xcpy = (int *)malloc(sizeof(int) * nparticles);
    int *ycpy = (int *)malloc(sizeof(int) * nparticles);
    float *wcpy = (float *)malloc(sizeof(float) * nparticles);
    memcpy(xcpy, particles_x, sizeof(int) * nparticles);
    memcpy(ycpy, particles_y, sizeof(int) * nparticles);
    memcpy(wcpy, particles_weights, sizeof(float) * nparticles);

    // Sort each array according to the sorted indices
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = wcpy[indices[i]];
        particles_x[i] = xcpy[indices[i]];
        particles_y[i] = ycpy[indices[i]];
    }

   free(xcpy);
   free(ycpy);
   free(wcpy);
   xcpy = nullptr;
   ycpy = nullptr;
   wcpy = nullptr;
}

static void normalize_weights(unsigned int nparticles, float *particles_weights)
{
    float sum = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        sum += particles_weights[i];
    }

    if (sum > 0.0)
    {
        for (unsigned int i = 0; i < nparticles; i++)
        {
            particles_weights[i] /= sum;
            assert((particles_weights[i] >= 0.0) && (particles_weights[i] <= 1.0));
        }
    }
}

static void complete_resample_and_move_step(unsigned int nparticles, float *particles_weights, std::mt19937 &rng, unsigned int *indices, int *particles_x, int *particles_y, int estimated_vx, int estimated_vy)
{
    // Create a distribution I will need
    auto dist = std::uniform_real_distribution<float>(0.0, 1.0);
    std::uniform_int_distribution<std::mt19937::result_type> height_distribution;
    std::uniform_int_distribution<std::mt19937::result_type> width_distribution;

    // Create the new particles in vectors
    std::vector<int> pxs;
    std::vector<int> pys;

    // Normalize the weights so that each one is between 0 and 1
    normalize_weights(nparticles, particles_weights);

    // Sort the particles by weight (in reverse - heaviest at the front of the array)
    sort_particles_by_weight_in_place(indices, nparticles, particles_weights, particles_x, particles_y);

    // Align a CMF (cumulative mass function) array, where each bin is the sum of all previous weights
    std::vector<float> cmf;
    float acc_prob_mass = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        acc_prob_mass += particles_weights[i];
        cmf.push_back(acc_prob_mass);
    }

    // Do a search into the CMF to find the place where our randomly generated probability (0 to 1) fits
    for (unsigned int i = 0; i < nparticles; i++)
    {
        float p = dist(rng);
        assert((p <= 1.0) && (p >= 0.0));

        int cmf_index = -1;
        for (unsigned int j = 0; j < nparticles; j++)
        {
            // Search for where the generated probability belongs
            if (p <= cmf[j])
            {
                cmf_index = j;
                break;
            }
        }

        if (cmf_index >= 0)
        {
            pxs.push_back(particles_x[cmf_index]);
            pys.push_back(particles_y[cmf_index]);
        }
        else
        {
            // Probabilities are all zero. Resample from uniform.
            pxs.push_back(width_distribution(rng));
            pys.push_back(height_distribution(rng));
        }
    }

    // Now overwrite the current batch of particles with the new ones
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_x[i] = pxs[i];
        particles_y[i] = pys[i];
    }

    // Move particles
    for (unsigned int i = 0; i < nparticles; i++)
    {
        static const float sigma = 2.5;
        float vx = gaussian_noise(estimated_vx, sigma, rng);
        float vy = gaussian_noise(estimated_vy, sigma, rng);
        particles_x[i] += vx;
        particles_y[i] += vy;
        particles_weights[i] = probability_of_value_from_bivariate_gaussian(vx, vy, estimated_vx, estimated_vy, sigma, sigma);
    }
}

static void resample_particles(unsigned int nparticles, float *particles_weights, std::mt19937 &rng, unsigned int *indices, int *particles_x, int *particles_y)
{
    // Create a distribution I will need
    auto dist = std::uniform_real_distribution<float>(0.0, 1.0);
    std::uniform_int_distribution<std::mt19937::result_type> height_distribution;
    std::uniform_int_distribution<std::mt19937::result_type> width_distribution;

    // Create the new particles in vectors
    std::vector<int> pxs;
    std::vector<int> pys;

    // Normalize the weights so that each one is between 0 and 1
    normalize_weights(nparticles, particles_weights);

    // Sort the particles by weight (in reverse - heaviest at the front of the array)
    sort_particles_by_weight_in_place(indices, nparticles, particles_weights, particles_x, particles_y);

    // Align a CMF (cumulative mass function) array, where each bin is the sum of all previous weights
    std::vector<float> cmf;
    float acc_prob_mass = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        acc_prob_mass += particles_weights[i];
        cmf.push_back(acc_prob_mass);
    }

    // Do a search into the CMF to find the place where our randomly generated probability (0 to 1) fits
    for (unsigned int i = 0; i < nparticles; i++)
    {
        float p = dist(rng);
        assert((p <= 1.0) && (p >= 0.0));

        int cmf_index = -1;
        for (unsigned int j = 0; j < nparticles; j++)
        {
            // Search for where the generated probability belongs
            if (p <= cmf[j])
            {
                cmf_index = j;
                break;
            }
        }

        if (cmf_index >= 0)
        {
            pxs.push_back(particles_x[cmf_index]);
            pys.push_back(particles_y[cmf_index]);
        }
        else
        {
            // Probabilities are all zero. Resample from uniform.
            pxs.push_back(width_distribution(rng));
            pys.push_back(height_distribution(rng));
        }
    }

    // Now overwrite the current batch of particles with the new ones
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_x[i] = pxs[i];
        particles_y[i] = pys[i];
    }
}

__global__ void kernel_normalize_weights_reduction(unsigned int nparticles, float *dev_weights, float *intermediate)
{
    // Dynamically-sized shared memory buffer for the reduction (this should be no smaller than blockDim.x)
    extern __shared__ float tmp[];

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // load all weights in this block into temp array
    if (index < nparticles)
    {
        tmp[index] = dev_weights[index];
    }
    __syncthreads();

    // Now do a binary sum tree to reduce to a single accumulated total weight
    for (int stride = 1; stride < nparticles; stride *= 2)
    {
        if ((index < nparticles) && ((index - stride) >= 0))
        {
            tmp[index] += tmp[index - stride];
        }
        __syncthreads();
    }

    // Each block now needs to add its total to its index in intermediate
    // So determine which thread should do this, since we only need one
    // item from each block
    bool lastusefulthread;
    if (blockIdx.x == (gridDim.x - 1))
    {
        // If my block index is that of the final block, then I am
        // the thread responsible for the last useful item if
        // my index is that of the final particle
        lastusefulthread = (index == (nparticles - 1));
    }
    else
    {
        // If my block is not the final one, then I am
        // the thread responsible for the last useful item if
        // my index is that of the final item in this block
        lastusefulthread = (threadIdx.x == (blockDim.x - 1));
    }

    if (lastusefulthread)
    {
        intermediate[blockIdx.x] = tmp[threadIdx.x];
    }
}

__global__ void kernel_normalize_weights_complete(unsigned int nparticles, float *dev_weights, float summed_weights)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Divide all weights by sum in parallel
    if ((index < nparticles) && summed_weights > 0.0f)
    {
        dev_weights[index] /= summed_weights;
    }
}

int device_resample_and_move(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, float *particles_weights, std::mt19937 &rng, unsigned int *indices, int nthreads_per_block)
{
    #if 1
    ///////////////////////////////////////////////////////////////////////////////////////////
    hipError_t err;
    int *dev_particles_x = nullptr;
    int *dev_particles_y = nullptr;
    float *dev_weights = nullptr;
    unsigned int *dev_indices = nullptr;
    float *dev_sum_tmp = nullptr;   // The temporary results from each block during sum
    float *sum_tmp = nullptr;
    float summed_weights = 0.0;
    int nblocks = ceil(nparticles / (float)nthreads_per_block);

    #define CHECK_CUDA_ERR(err) do { if (err != hipSuccess) { err = (hipError_t)__LINE__; goto fail; }} while (0)

    /* Allocate everything we need */
    err = hipMalloc(&dev_particles_x, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_particles_y, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_weights, nparticles * sizeof(float));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_indices, nparticles * sizeof(unsigned int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_sum_tmp, nblocks * sizeof(float));
    CHECK_CUDA_ERR(err);

    /* Copy everything to the device */
    err = hipMemcpy(dev_particles_x, particles_x, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_particles_y, particles_y, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_weights, particles_weights, nparticles * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_indices, indices, nparticles * sizeof(unsigned int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    ///* Launch kernels */
    //kernel_normalize_weights_reduction<<<nblocks, nthreads_per_block, (sizeof(float) * nthreads_per_block)>>>(nparticles, dev_weights, dev_sum_tmp);
    //err = hipDeviceSynchronize();
    //CHECK_CUDA_ERR(err);

    //// Sequential sum of the intermediate results in dev_sum_tmp
    //sum_tmp = (float *)malloc(nblocks * sizeof(float));
    //err = hipMemcpy(sum_tmp, dev_sum_tmp, nblocks * sizeof(float), hipMemcpyDeviceToHost);
    //CHECK_CUDA_ERR(err);
    //for (unsigned int i = 0; i < nblocks; i++)
    //{
    //    summed_weights += sum_tmp[i];
    //}
    //free(sum_tmp);
    //sum_tmp = nullptr;

    //kernel_normalize_weights_complete<<<nblocks, nthreads_per_block>>>(nparticles, dev_weights, summed_weights);
    //err = hipDeviceSynchronize();
    //CHECK_CUDA_ERR(err);

    //kernel_sort_particles
    //kernel_resample_particles
    //kernel_reset_all_weights
    //kernel_move_particles

    //&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&//
    //Remove the logic here as you convert it to CUDA
    //&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&//
    complete_resample_and_move_step(nparticles, particles_weights, rng, indices, particles_x, particles_y, estimated_vx, estimated_vy);
    //&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&//
    // End
    //&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&//

    /* Transfer results back to host */
    err = hipMemcpy(particles_x, dev_particles_x, nparticles * sizeof(int), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(particles_y, dev_particles_y, nparticles * sizeof(int), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(particles_weights, dev_weights, nparticles * sizeof(float), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(indices, dev_indices, nparticles * sizeof(unsigned int), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    /* Free up memory */
    err = hipFree(dev_particles_x);
    dev_particles_x = nullptr;
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_particles_y);
    dev_particles_y = nullptr;
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_weights);
    dev_weights = nullptr;
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_indices);
    dev_indices = nullptr;
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_sum_tmp);
    dev_sum_tmp = nullptr;
    CHECK_CUDA_ERR(err);

    #undef CHECK_CUDA_ERR

fail:
    assert(err == hipSuccess);
    return err;
    ///////////////////////////////////////////////////////////////////////////////////////////
#else
    // Resample from weights
    resample_particles(nparticles, particles_weights, rng, indices, particles_x, particles_y);

    // Reset all weights
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = 0.0;
    }

    // Move all particles according to our movement model (plus Gaussian noise)
    // Also update weights based on how likely the movements were
    move_particles(estimated_vx, estimated_vy, nparticles, particles_x, particles_y, particles_weights, rng);

    return 0;
#endif
}

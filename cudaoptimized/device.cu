#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <random>
#include <vector>
#include "device.h"

static double gaussian_noise(double mean, double sigma, std::mt19937 &rng)
{
    std::normal_distribution<double> gaussian(mean, sigma);
    return gaussian(rng);
}

static double probability_of_value_from_bivariate_gaussian(double x, double y, double mean_x, double mean_y, double sigma_x, double sigma_y)
{
    const double rho = 0.0; // cov / (sig1 * sig2); Covariance of two independent random variables is zero.
    double denom = 2.0 * M_PI * sigma_x * sigma_y * sqrt(1.0 - (rho * rho));
    double A = ((x - mean_x) * (x - mean_x)) / (sigma_x * sigma_x);
    double B = ((2.0 * rho * (x - mean_x) * (y - mean_y)) / (sigma_x * sigma_y));
    double C = ((y - mean_y) * (y - mean_y)) / (sigma_y * sigma_y);
    A /= 1000.0;  // For numerical stability
    C /= 1000.0;  // Ditto
    double z = A - B + C;
    double a = (-1.0 * z) / (2.0 * (1.0 - rho * rho));

    return exp(a) / denom;
}

void device_calculate_likelihood(const int *particles_x, const int *particles_y, int estimate_x, int estimate_y, double *weights, unsigned int nparticles)
{
    /*
        P(A | B) = P(B | A) * P(A)   /  P(B)
        P(location | measurement) = P(measurement | location) * P(location) / P(measurement)

        We can actually just use: P(measurement | lcoation) * P(location), discarding the probability of
        the measurement, since all particles are using the same measurement, and all I care about is
        the relative probability of each particle, not the true probability.

        So, using:
        P(measurement | location) = Gaussian(mean=location, std=who knows)
        P(location) = Uniform probability over the whole range (we don't have a reason to believe one location
                      is more probable in general than any other).

        Since P(location) is uniform, and therefore doesn't matter per particle, we can also do away with it,
        leaving just a Gaussian with mean centered on the location.

        So now we need the probability of measured_x and measured_y, given a Gaussian around location.
    */
    for (unsigned int i = 0; i < nparticles; i++)
    {
        double x = (double)particles_x[i];
        double y = (double)particles_y[i];
        weights[i] = probability_of_value_from_bivariate_gaussian(x, y, estimate_x, estimate_y, 2.5, 2.5);
    }
}

static void move_particles(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, double *particles_weights, std::mt19937 &rng)
{
    for (unsigned int i = 0; i < nparticles; i++)
    {
        static const double sigma = 2.5;
        double vx = gaussian_noise(estimated_vx, sigma, rng);
        double vy = gaussian_noise(estimated_vy, sigma, rng);
        particles_x[i] += vx;
        particles_y[i] += vy;
        particles_weights[i] = probability_of_value_from_bivariate_gaussian(vx, vy, estimated_vx, estimated_vy, sigma, sigma);
    }
}

static void sort_particles_by_weight_in_place(unsigned int *indices, unsigned int nparticles, double *particles_weights, int *particles_x, int *particles_y)
{
    // Sort the indices
    std::sort(indices, indices + nparticles, SortIndices(particles_weights));

    // Make copies of the three arrays (gross)
    int *xcpy = (int *)malloc(sizeof(int) * nparticles);
    int *ycpy = (int *)malloc(sizeof(int) * nparticles);
    double *wcpy = (double *)malloc(sizeof(double) * nparticles);
    memcpy(xcpy, particles_x, sizeof(int) * nparticles);
    memcpy(ycpy, particles_y, sizeof(int) * nparticles);
    memcpy(wcpy, particles_weights, sizeof(double) * nparticles);

    // Sort each array according to the sorted indices
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = wcpy[indices[i]];
        particles_x[i] = xcpy[indices[i]];
        particles_y[i] = ycpy[indices[i]];
    }

   free(xcpy);
   free(ycpy);
   free(wcpy);
   xcpy = nullptr;
   ycpy = nullptr;
   wcpy = nullptr;
}

static void normalize_weights(unsigned int nparticles, double *particles_weights)
{
    double sum = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        sum += particles_weights[i];
    }

    if (sum > 0.0)
    {
        for (unsigned int i = 0; i < nparticles; i++)
        {
            particles_weights[i] /= sum;
            assert((particles_weights[i] >= 0.0) && (particles_weights[i] <= 1.0));
        }
    }
}

static void resample_particles(unsigned int nparticles, double *particles_weights, std::mt19937 &rng, unsigned int *indices, int *particles_x, int *particles_y)
{
    // Create a distribution I will need
    auto dist = std::uniform_real_distribution<double>(0.0, 1.0);
    std::uniform_int_distribution<std::mt19937::result_type> height_distribution;
    std::uniform_int_distribution<std::mt19937::result_type> width_distribution;

    // Create the new particles in vectors
    std::vector<int> pxs;
    std::vector<int> pys;

    // Normalize the weights so that each one is between 0 and 1
    normalize_weights(nparticles, particles_weights);

    // Sort the particles by weight (in reverse - heaviest at the front of the array)
    sort_particles_by_weight_in_place(indices, nparticles, particles_weights, particles_x, particles_y);

    // Align a CMF (cumulative mass function) array, where each bin is the sum of all previous weights
    std::vector<double> cmf;
    double acc_prob_mass = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        acc_prob_mass += particles_weights[i];
        cmf.push_back(acc_prob_mass);
    }

    // Do a search into the CMF to find the place where our randomly generated probability (0 to 1) fits
    for (unsigned int i = 0; i < nparticles; i++)
    {
        double p = dist(rng);
        assert((p <= 1.0) && (p >= 0.0));

        int cmf_index = -1;
        for (unsigned int j = 0; j < nparticles; j++)
        {
            // Search for where the generated probability belongs
            if (p <= cmf[j])
            {
                cmf_index = j;
                break;
            }
        }

        if (cmf_index >= 0)
        {
            pxs.push_back(particles_x[cmf_index]);
            pys.push_back(particles_y[cmf_index]);
        }
        else
        {
            // Probabilities are all zero. Resample from uniform.
            pxs.push_back(width_distribution(rng));
            pys.push_back(height_distribution(rng));
        }
    }

    // Now overwrite the current batch of particles with the new ones
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_x[i] = pxs[i];
        particles_y[i] = pys[i];
    }
}

void device_resample_and_move(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, double *particles_weights, std::mt19937 &rng, unsigned int *indices)
{
    // Resample from weights
    resample_particles(nparticles, particles_weights, rng, indices, particles_x, particles_y);

    // Reset all weights
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = 0.0;
    }

    // Move all particles according to our movement model (plus Gaussian noise)
    // Also update weights based on how likely the movements were
    move_particles(estimated_vx, estimated_vy, nparticles, particles_x, particles_y, particles_weights, rng);
}

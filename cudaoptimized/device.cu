#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <random>
#include <vector>
#include "device.h"

static float gaussian_noise(float mean, float sigma, std::mt19937 &rng)
{
    std::normal_distribution<float> gaussian(mean, sigma);
    return gaussian(rng);
}

static float probability_of_value_from_bivariate_gaussian(float x, float y, float mean_x, float mean_y, float sigma_x, float sigma_y)
{
    const float rho = 0.0; // cov / (sig1 * sig2); Covariance of two independent random variables is zero.
    float denom = 2.0 * M_PI * sigma_x * sigma_y * sqrt(1.0 - (rho * rho));
    float A = ((x - mean_x) * (x - mean_x)) / (sigma_x * sigma_x);
    float B = ((2.0 * rho * (x - mean_x) * (y - mean_y)) / (sigma_x * sigma_y));
    float C = ((y - mean_y) * (y - mean_y)) / (sigma_y * sigma_y);
    A /= 1000.0;  // For numerical stability
    C /= 1000.0;  // Ditto
    float z = A - B + C;
    float a = (-1.0 * z) / (2.0 * (1.0 - rho * rho));

    return exp(a) / denom;
}

__global__ void kernel_calculate_likelihood(int *particles_x, int *particles_y, float *weights, unsigned int nparticles, float estimate_x, float estimate_y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nparticles)
    {
        float x = particles_x[index];
        float y = particles_y[index];

        const float sigma_x = 2.5;
        const float sigma_y = 2.5;
        float mean_x = estimate_x;
        float mean_y = estimate_y;

        // Compute the probability of getting this x,y combo from a distribution centered at estimate_x, estimte_y.
        const float rho = 0.0; // cov / (sig1 * sig2); Covariance of two independent random variables is zero.
        float denom = 2.0f * M_PI * sigma_x * sigma_y * sqrt(1.0f - (rho * rho));
        float A = ((x - mean_x) * (x - mean_x)) / (sigma_x * sigma_x);
        float B = ((2.0f * rho * (x - mean_x) * (y - mean_y)) / (sigma_x * sigma_y));
        float C = ((y - mean_y) * (y - mean_y)) / (sigma_y * sigma_y);
        A /= 1000.0f;  // For numerical stability
        C /= 1000.0f;  // Ditto
        float z = A - B + C;
        float a = (-1.0f * z) / (2.0f * (1.0f - rho * rho));

        float prob = exp(a) / denom;

        weights[index] = prob;
    }
}

int device_calculate_likelihood(const int *particles_x, const int *particles_y, int estimate_x, int estimate_y, float *weights, unsigned int nparticles)
{
#define OPTIMIZED 1
#if OPTIMIZED
    hipError_t err;
    int *dev_particles_x = nullptr;
    int *dev_particles_y = nullptr;
    float *dev_weights = nullptr;

    #define CHECK_CUDA_ERR(err) do { if (err != hipSuccess) { err = (hipError_t)__LINE__; goto fail; }} while (0)

    /* Malloc all the device memory we need */
    err = hipMalloc(&dev_particles_x, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_particles_y, nparticles * sizeof(int));
    CHECK_CUDA_ERR(err);

    err = hipMalloc(&dev_weights, nparticles * sizeof(float));
    CHECK_CUDA_ERR(err);

    /* Copy arrays onto device */
    err = hipMemcpy(dev_particles_x, particles_x, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_particles_y, particles_y, nparticles * sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    err = hipMemcpy(dev_weights, weights, nparticles * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERR(err);

    /* Call the kernel */
    kernel_calculate_likelihood<<<ceil(nparticles / 512.0), 512>>>(dev_particles_x, dev_particles_y, dev_weights, estimate_x, estimate_y, nparticles);

    /* Copy array back onto host */
    err = hipMemcpy(weights, dev_weights, nparticles * sizeof(float), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERR(err);

    /* Deallocate the device arrays */
    err = hipFree(dev_particles_x);
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_particles_y);
    CHECK_CUDA_ERR(err);

    err = hipFree(dev_weights);
    CHECK_CUDA_ERR(err);

    #undef CHECK_CUDA_ERR

fail:
    assert(err == hipSuccess);
    return (int)err;
#else

    for (unsigned int i = 0; i < nparticles; i++)
    {
        float x = (float)particles_x[i];
        float y = (float)particles_y[i];

        weights[i] = probability_of_value_from_bivariate_gaussian(x, y, estimate_x, estimate_y, 2.5, 2.5);
    }

    return 0;
#endif
}

static void move_particles(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, float *particles_weights, std::mt19937 &rng)
{
    for (unsigned int i = 0; i < nparticles; i++)
    {
        static const float sigma = 2.5;
        float vx = gaussian_noise(estimated_vx, sigma, rng);
        float vy = gaussian_noise(estimated_vy, sigma, rng);
        particles_x[i] += vx;
        particles_y[i] += vy;
        particles_weights[i] = probability_of_value_from_bivariate_gaussian(vx, vy, estimated_vx, estimated_vy, sigma, sigma);
    }
}

static void sort_particles_by_weight_in_place(unsigned int *indices, unsigned int nparticles, float *particles_weights, int *particles_x, int *particles_y)
{
    // Sort the indices
    std::sort(indices, indices + nparticles, SortIndices(particles_weights));

    // Make copies of the three arrays (gross)
    int *xcpy = (int *)malloc(sizeof(int) * nparticles);
    int *ycpy = (int *)malloc(sizeof(int) * nparticles);
    float *wcpy = (float *)malloc(sizeof(float) * nparticles);
    memcpy(xcpy, particles_x, sizeof(int) * nparticles);
    memcpy(ycpy, particles_y, sizeof(int) * nparticles);
    memcpy(wcpy, particles_weights, sizeof(float) * nparticles);

    // Sort each array according to the sorted indices
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = wcpy[indices[i]];
        particles_x[i] = xcpy[indices[i]];
        particles_y[i] = ycpy[indices[i]];
    }

   free(xcpy);
   free(ycpy);
   free(wcpy);
   xcpy = nullptr;
   ycpy = nullptr;
   wcpy = nullptr;
}

static void normalize_weights(unsigned int nparticles, float *particles_weights)
{
    float sum = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        sum += particles_weights[i];
    }

    if (sum > 0.0)
    {
        for (unsigned int i = 0; i < nparticles; i++)
        {
            particles_weights[i] /= sum;
            assert((particles_weights[i] >= 0.0) && (particles_weights[i] <= 1.0));
        }
    }
}

static void resample_particles(unsigned int nparticles, float *particles_weights, std::mt19937 &rng, unsigned int *indices, int *particles_x, int *particles_y)
{
    // Create a distribution I will need
    auto dist = std::uniform_real_distribution<float>(0.0, 1.0);
    std::uniform_int_distribution<std::mt19937::result_type> height_distribution;
    std::uniform_int_distribution<std::mt19937::result_type> width_distribution;

    // Create the new particles in vectors
    std::vector<int> pxs;
    std::vector<int> pys;

    // Normalize the weights so that each one is between 0 and 1
    normalize_weights(nparticles, particles_weights);

    // Sort the particles by weight (in reverse - heaviest at the front of the array)
    sort_particles_by_weight_in_place(indices, nparticles, particles_weights, particles_x, particles_y);

    // Align a CMF (cumulative mass function) array, where each bin is the sum of all previous weights
    std::vector<float> cmf;
    float acc_prob_mass = 0.0;
    for (unsigned int i = 0; i < nparticles; i++)
    {
        acc_prob_mass += particles_weights[i];
        cmf.push_back(acc_prob_mass);
    }

    // Do a search into the CMF to find the place where our randomly generated probability (0 to 1) fits
    for (unsigned int i = 0; i < nparticles; i++)
    {
        float p = dist(rng);
        assert((p <= 1.0) && (p >= 0.0));

        int cmf_index = -1;
        for (unsigned int j = 0; j < nparticles; j++)
        {
            // Search for where the generated probability belongs
            if (p <= cmf[j])
            {
                cmf_index = j;
                break;
            }
        }

        if (cmf_index >= 0)
        {
            pxs.push_back(particles_x[cmf_index]);
            pys.push_back(particles_y[cmf_index]);
        }
        else
        {
            // Probabilities are all zero. Resample from uniform.
            pxs.push_back(width_distribution(rng));
            pys.push_back(height_distribution(rng));
        }
    }

    // Now overwrite the current batch of particles with the new ones
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_x[i] = pxs[i];
        particles_y[i] = pys[i];
    }
}

int device_resample_and_move(int estimated_vx, int estimated_vy, unsigned int nparticles, int *particles_x, int *particles_y, float *particles_weights, std::mt19937 &rng, unsigned int *indices)
{
    // Resample from weights
    resample_particles(nparticles, particles_weights, rng, indices, particles_x, particles_y);

    // Reset all weights
    for (unsigned int i = 0; i < nparticles; i++)
    {
        particles_weights[i] = 0.0;
    }

    // Move all particles according to our movement model (plus Gaussian noise)
    // Also update weights based on how likely the movements were
    move_particles(estimated_vx, estimated_vy, nparticles, particles_x, particles_y, particles_weights, rng);

    return 0;
}
